#include "hip/hip_runtime.h"
//
// Created by arobot on 7/7/2024.
//
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <hip/hip_runtime.h>

___global__ void vectorAdd(int* a, int* b, int* c, int size) {
    int i = threadIdx.x; // the .x value is the index of the thread
    if (i < size) {
        c[i] = a[i] + b[i];
        return;
    }
}


int main() {
    int a[] = {1, 2, 3, 4, 5};
    int b[] = {1, 2, 3, 4, 5};

    int c[sizeof(a) /sizeof(int)] = { 0 }; // array a and b are stored in array c

    // marks: the cpu calculation
    for (int i = 0; i < sizeof(c) / sizeof(int); i++) {
        c[i] = a[i] + b[i];
    }


    // marks: the sttart of GPU processing
    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    // allocate memory intoo the gpu by by creatin  pointers so  pointers into the gpu
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    //coopy the vectors into the  gpu --> puts the data into the gpu for processing
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
    //call to the function. must use <<< and 'GRID_SIZE' and 'BLOCK_SIZE' to call the function
    // IE-> vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>
    // GRID_SIZE is the number of blocks to use
    // BLOCK_SIZE is the number of threads to use

    // SET TO ONE BLOCK AND IS THE SIZE OF THE ARRAY (5)
    vectorAdd<<<1, sizeof(c) / sizeof(int)>>>(cudaA, cudaB, cudaC, sizeof(c) / sizeof(int));
    hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice); // copies the data out of c

    return 0;
}