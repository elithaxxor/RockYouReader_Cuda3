#include "hip/hip_runtime.h"



#include <ctime>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
// Theads are organized in blocks in GPUS (since they have so much more threads than CPUs)
// The blocks are organized in a grid (a 3D array of blocks)
//  This is a test to see if the GPU can add two arrays together

using namespace std;
int count = 1000; // number of elements in the array


// THE VARIABLE "ID" IS GOING TO BE A UNINQUE INDEX OF THE THREAD, SO MANY THREADS CAN RUN AT THE SAME TIME
__global__ void vectorAdd(int* a, int* b, int* c, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // the .x value is the index of the thread
	printf(" the id is %d\n", id, "\n", "The size is %d\n", size, "\n\n" Block Index X: "%d\n", blockIdx.x, "\n", "Thread Index X: %d\n", threadIdx.x, "\n\n");
    if (i < size) {
        c[i] = a[i] + b[i];
        return;
    }
}
int main2() {

    srand(time(NULL));
    int *arrA = new int [count];
    int *arrB = new int [count];


    // MARK: the array is filled with random numbers (0-1000)
    for (int  i = 0 ; i < count; i++) {
        arrA[i] = rand() % 1000;
        arrB[i] = rand() % 1000;
    }

    printf("5 elements of the arrays, prior to addtion(running CPU calculation)", h_a, "\n", h_b);
    for(int i = 0; i < count; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], "\n",  h_a[i] + h_b[i]);
    }

    // MARK: Check if memory allocation condionts are met
    if(hipMalloc(d_a, sizeof(int) * count) != hipSuccess) {
        printf("Error allocating memory for d_a\n");
        hipFree(d_b);
        return 1;
    }
    if(hipMalloc(&d_b, sizeof(int) * count) != hipSuccess) {
        printf("Error allocating memory for d_b\n");
        hipFree(d_a);
        return 1;
    }

    // Copys the value from the hosts calculation to the GPU
    if(hipMemcpy(d_a, arrA, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying memory to d_a\n");
        hipFree(d_a);
        hipFree(d_b);
        return 1;
    }
    if(hipMemcpy(d_b, arrB, sizeof(int) * count, hipMemcpyHostToDevice) != hipSuccess) {
        printf("Error copying memory to d_b\n");
        hipFree(d_a);
        hipFree(d_b);
        return 1;
    }

    // MARK: The GPU calculation, calls the kernal
    //->> this is where  the threads per block and the number of blocks are set
    vectorAdd<<<count / 256 + 1, 256>>(d_a, d_b, d_c, count);

    if(hipGetLastError() != hipSuccess) {
        printf("Error launching the kernal\n");
        hipFree(d_a);
        hipFree(d_b);
        delete[] arrA;
        delete[] arrB;
        return 1;
    }

    // Checks if GPU calculation was successful by comparing to CPU calculation
    print("5 elements of the arrays, after addtion(running GPU calculation)\n");
    for(int i = 0, i < count; i++) {
        print("%d + %d = %d\n", arrA[i], arrB[i], arrC[i]);
    }

    hipFree(d_a);
    hipFree(d_b);
    delete[] arrA;
    delete[] arrB;

}


return 0 ;
}
